#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : EM_gpu.cu
 Author      : Jiaxiao Wu
 Version     :
 Copyright   : 
 Description : CUDA expectation-maximization algorithm
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "EM_gpu.h"
extern "C" {
#include "matrix.h"
}

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);

#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)
#define WARPSIZE 32
#define THR_PER_BLOCK 256
#define BLOCKSIZE_S 64

#define MATH_PI 3.14159265358979323846   // pi


/**
 * CUDA kernels
 */
__global__ void set_zeros_kernel (double *data, int height, int width)
{
	int idx = blockIdx.x*blockDim.x+threadIdx.x;
	if (idx < height * width)
		data[idx] = 0;
}

__global__ void marginal_single_reduction(double *g_idata, double *g_odata)
{
	extern __shared__ double sdata[];
	int tid = threadIdx.x;

	// Read data into shared memory
	sdata[tid] = g_idata[tid];
	__syncthreads();

	// Reduction in shared memory
	for (int s = blockDim.x / 2; s > WARPSIZE; s>>=1)    // Reversed loop
	{
		if (tid < s)                 // Non-divergent branch
			sdata[tid] += sdata[tid + s]; // Sequential addressing
		__syncthreads();
	}

	if (tid < WARPSIZE)
	{
		if (tid >= 64) sdata[tid] += sdata[tid + 32];
		if (tid >= 32) sdata[tid] += sdata[tid + 16];
		if (tid >= 16) sdata[tid] += sdata[tid + 8];
		if (tid >= 8) sdata[tid] += sdata[tid + 4];
		if (tid >= 4) sdata[tid] += sdata[tid + 2];
		if (tid >= 2) sdata[tid] += sdata[tid + 1];
	}

	// Write the sum of this block to device memory
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__global__ void marginal_reduction(double *likelihood, double *tmp_odata, int size, int num_gaus)
{
	int tid = threadIdx.x;
	int lx = blockDim.x * blockIdx.x + threadIdx.x;
	int ly = blockIdx.y;

	if (lx < size && ly < num_gaus)
	{
		// Read data into shared memory
		extern __shared__ double sdata[];
		sdata[tid] = likelihood[ly * size + lx];
		__syncthreads();

		// Reduction in shared memory
		for (int s = blockDim.x / 2; s > WARPSIZE; s>>=1)    // Reversed loop
		{
			if (tid < s)                 // Non-divergent branch
				sdata[tid] += sdata[tid + s]; // Sequential addressing
			__syncthreads();
		}

		if (tid < WARPSIZE)
		{
			if (tid >= 64) sdata[tid] += sdata[tid + 32];
			if (tid >= 32) sdata[tid] += sdata[tid + 16];
			if (tid >= 16) sdata[tid] += sdata[tid + 8];
			if (tid >= 8) sdata[tid] += sdata[tid + 4];
			if (tid >= 4) sdata[tid] += sdata[tid + 2];
			if (tid >= 2) sdata[tid] += sdata[tid + 1];
		}

		// Write the sum of this block to device memory
		if (tid == 0) tmp_odata[blockIdx.y * gridDim.x + blockIdx.x] = sdata[0];
	}
}

__global__ void mu_reduction(double *likelihood, double *samples, double *marg, double *tmp_odata, int size, int num_gaus, int dim)
{
	int tid = threadIdx.x;
	int blockId = (gridDim.x * gridDim.y) * blockIdx.z + gridDim.x * blockIdx.y + blockIdx.x;

	int samp_idx = blockIdx.x * blockDim.x + threadIdx.x;
	int dim_idx = blockIdx.y;
	int gaus_idx = blockIdx.z;

	if (samp_idx < size && dim_idx < dim && gaus_idx < num_gaus)
	{
		// Read data into shared memory
		extern __shared__ double sdata[];
		sdata[tid] = likelihood[gaus_idx * size + samp_idx] * samples[samp_idx * dim + dim_idx];
		__syncthreads();

		// Reduction in shared memory
		for (int s = blockDim.x / 2; s > WARPSIZE; s>>=1)    // Reversed loop
		{
			if (tid < s)                      // Non-divergent branch
				sdata[tid] += sdata[tid + s]; // Sequential addressing
			__syncthreads();
		}

		if (tid < WARPSIZE)
		{
			if (tid >= 64) sdata[tid] += sdata[tid + 32];
			if (tid >= 32) sdata[tid] += sdata[tid + 16];
			if (tid >= 16) sdata[tid] += sdata[tid + 8];
			if (tid >= 8) sdata[tid] += sdata[tid + 4];
			if (tid >= 4) sdata[tid] += sdata[tid + 2];
			if (tid >= 2) sdata[tid] += sdata[tid + 1];
		}

		// Write the sum of this block to device memory
		if (tid == 0) tmp_odata[blockId] = sdata[0] / (double) marg[gaus_idx];
	}
}

__global__ void sigma_reduction(double *likelihood, double *samples, double *marg, double *mu, double *odata,
		int size, int num_gaus, int dim)
{
	int tid = threadIdx.x;
	int blockId = (gridDim.x * gridDim.y) * blockIdx.z + gridDim.x * blockIdx.y + blockIdx.x;

	int samp_idx = blockIdx.x * blockDim.x + threadIdx.x;
	int gaus_idx = blockIdx.z;

	int sigy = blockIdx.y / dim;
	int sigx = blockIdx.y - dim * sigy;

	if (samp_idx < size && gaus_idx < num_gaus && sigx < dim && sigy < dim)
	{
		// Read data into shared memory
		extern __shared__ double sdata[];

		sdata[tid] = likelihood[gaus_idx * size + samp_idx]
		                     * (samples[samp_idx * dim + sigx] - mu[gaus_idx * dim + sigx]) // non-coalesced access
		                     * (samples[samp_idx * dim + sigy] - mu[gaus_idx * dim + sigy]);
		__syncthreads();

		// Reduction in shared memory
		for (int s = blockDim.x / 2; s > WARPSIZE; s>>=1)    // Reversed loop
		{
			if (tid < s)                 // Non-divergent branch
				sdata[tid] += sdata[tid + s]; // Sequential addressing
			__syncthreads();
		}

		if (tid < WARPSIZE)
		{
			if (tid >= 64) sdata[tid] += sdata[tid + 32];
			if (tid >= 32) sdata[tid] += sdata[tid + 16];
			if (tid >= 16) sdata[tid] += sdata[tid + 8];
			if (tid >= 8) sdata[tid] += sdata[tid + 4];
			if (tid >= 4) sdata[tid] += sdata[tid + 2];
			if (tid >= 2) sdata[tid] += sdata[tid + 1];
		}

		// Write the sum of this block to device memory
		if (tid == 0) odata[blockId] = sdata[0] / (double) marg[gaus_idx];
	}
}

__global__ void single_reduction(double *idata, double *omat)
{
	extern __shared__ double sdata[];
	int tid = threadIdx.x;
	int gid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

	// Read data into shared memory
	sdata[tid] = idata[gid];
	__syncthreads();

	// Reduction in shared memory
	for (int s = blockDim.x / 2; s > WARPSIZE; s>>=1)    // Reversed loop
	{
		if (tid < s)                 // Non-divergent branch
			sdata[tid] += sdata[tid + s]; // Sequential addressing
		__syncthreads();
	}

	if (tid < WARPSIZE)
	{
		if (tid >= 64) sdata[tid] += sdata[tid + 32];
		if (tid >= 32) sdata[tid] += sdata[tid + 16];
		if (tid >= 16) sdata[tid] += sdata[tid + 8];
		if (tid >= 8) sdata[tid] += sdata[tid + 4];
		if (tid >= 4) sdata[tid] += sdata[tid + 2];
		if (tid >= 2) sdata[tid] += sdata[tid + 1];
	}

	// Write the sum of this block to device memory
	if (tid == 0) omat[blockIdx.y * gridDim.x + blockIdx.x] = sdata[0];
}

__global__ void mvnpdf_dim3(double *likelihood, double *samples, double *mu_mat, double *sig_mat, int size, int num_gaus)
{
	const int dim = 3;
	int lx = blockDim.x * blockIdx.x + threadIdx.x;
	int ly = blockIdx.y;
	if (lx < size && ly < num_gaus)
	{
		int i;
		__shared__ double x[dim];
		__shared__ double mu[dim];
		__shared__ double sigma[dim * dim];
		__shared__ double inv_sig[dim * dim];
		__shared__ double d[dim];

		// Read data from global to shared mem
		for (i = 0; i < dim; ++i)
		{
			x[i] = samples[lx * dim + i];
			mu[i] = mu_mat[ly * dim + i];
		}
		for (i = 0; i < dim * dim; ++i) sigma[i] = sig_mat[ly * dim * dim + i];


		// determinant and inverse of sigma
		double det = sigma[0] * (sigma[4] * sigma[8] - sigma[5] * sigma[7])
				- sigma[1] * (sigma[3] * sigma[8] - sigma[5] * sigma[6])
				+ sigma[2] * (sigma[3] * sigma[7] - sigma[4] * sigma[6]);

		inv_sig[0] = (sigma[4] * sigma[8] - sigma[5] * sigma[7]) / (double) det;
		inv_sig[1] = (sigma[2] * sigma[7] - sigma[1] * sigma[8]) / (double) det;
		inv_sig[2] = (sigma[1] * sigma[5] - sigma[2] * sigma[4]) / (double) det;
		inv_sig[3] = (sigma[5] * sigma[6] - sigma[3] * sigma[8]) / (double) det;
		inv_sig[4] = (sigma[0] * sigma[8] - sigma[2] * sigma[6]) / (double) det;
		inv_sig[5] = (sigma[2] * sigma[3] - sigma[0] * sigma[5]) / (double) det;
		inv_sig[6] = (sigma[3] * sigma[7] - sigma[4] * sigma[6]) / (double) det;
		inv_sig[7] = (sigma[1] * sigma[6] - sigma[0] * sigma[7]) / (double) det;
		inv_sig[8] = (sigma[0] * sigma[4] - sigma[1] * sigma[3]) / (double) det;

		// diff = x[i] - mu[i]
		for (i = 0; i < dim; ++i) d[i] = x[i] - mu[i];

		// expon = -1/2 * (x - mu)' * inv_sig * (x - mu)
		double expon = ((d[0]*inv_sig[0] + d[1]*inv_sig[3] + d[2]*inv_sig[6]) * d[0]
                      + (d[0]*inv_sig[1] + d[1]*inv_sig[4] + d[2]*inv_sig[7]) * d[1]
                      + (d[0]*inv_sig[2] + d[1]*inv_sig[5] + d[2]*inv_sig[8]) * d[2]) / (double) -2;

		// denom = sqrt((2pi)^dim * det(sigma))
		double denom = sqrt(pow(2 * MATH_PI, dim) * det);

		likelihood[ly * size + lx] = exp(expon) / (double) denom;
	}
	else likelihood[ly * size + lx] = 0;
}

__global__ void w_mvnpdf_dim3(double *likelihood, double *samples, double *mu_mat, double *sig_mat,
		double *weights, int size, int num_gaus)
{
	const int dim = 3;
	int lx = blockDim.x * blockIdx.x + threadIdx.x;
	int ly = blockIdx.y;
	if (lx < size && ly < num_gaus)
	{
		int i;
		__shared__ double x[dim];
		__shared__ double mu[dim];
		__shared__ double sigma[dim * dim];
		__shared__ double inv_sig[dim * dim];
		__shared__ double d[dim];
		double weight;

		// Read data from global to shared mem
		for (i = 0; i < dim; ++i)
		{
			x[i] = samples[lx * dim + i];
			mu[i] = mu_mat[ly * dim + i];
		}
		for (i = 0; i < dim * dim; ++i) sigma[i] = sig_mat[ly * dim * dim + i];
		weight = weights[ly];

		// determinant and inverse of sigma
		double det = sigma[0] * (sigma[4] * sigma[8] - sigma[5] * sigma[7])
				- sigma[1] * (sigma[3] * sigma[8] - sigma[5] * sigma[6])
				+ sigma[2] * (sigma[3] * sigma[7] - sigma[4] * sigma[6]);

		inv_sig[0] = (sigma[4] * sigma[8] - sigma[5] * sigma[7]) / (double) det;
		inv_sig[1] = (sigma[2] * sigma[7] - sigma[1] * sigma[8]) / (double) det;
		inv_sig[2] = (sigma[1] * sigma[5] - sigma[2] * sigma[4]) / (double) det;
		inv_sig[3] = (sigma[5] * sigma[6] - sigma[3] * sigma[8]) / (double) det;
		inv_sig[4] = (sigma[0] * sigma[8] - sigma[2] * sigma[6]) / (double) det;
		inv_sig[5] = (sigma[2] * sigma[3] - sigma[0] * sigma[5]) / (double) det;
		inv_sig[6] = (sigma[3] * sigma[7] - sigma[4] * sigma[6]) / (double) det;
		inv_sig[7] = (sigma[1] * sigma[6] - sigma[0] * sigma[7]) / (double) det;
		inv_sig[8] = (sigma[0] * sigma[4] - sigma[1] * sigma[3]) / (double) det;

		// diff = x[i] - mu[i]
		for (i = 0; i < dim; ++i) d[i] = x[i] - mu[i];

		// expon = -1/2 * (x - mu)' * inv_sig * (x - mu)
		double expon = ((d[0]*inv_sig[0] + d[1]*inv_sig[3] + d[2]*inv_sig[6]) * d[0]
                      + (d[0]*inv_sig[1] + d[1]*inv_sig[4] + d[2]*inv_sig[7]) * d[1]
                      + (d[0]*inv_sig[2] + d[1]*inv_sig[5] + d[2]*inv_sig[8]) * d[2]) / (double) -2;

		// denom = sqrt((2pi)^dim * det(sigma))
		double denom = sqrt(pow(2 * MATH_PI, dim) * det);

		likelihood[ly * size + lx] = weight * exp(expon) / (double) denom;
	}
	else likelihood[ly * size + lx] = 0;
}

__global__ void normalization(double *likelihood, int size, int num_gaus)
{
	int tid = threadIdx.y * blockDim.x + threadIdx.x;
	int lx = blockDim.x * blockIdx.x + threadIdx.x;
	int ly = threadIdx.y;

	if (lx < size && ly < num_gaus)
	{
		int i;
		double sum = 0;
		extern __shared__ double sdata[];

		// read data from global to shared mem
		sdata[tid] = likelihood[ly * size + lx];
		__syncthreads();

		// sum up likelihood values for the same sample
		for (i = 0; i < num_gaus; ++i)
			sum = sum + sdata[i * blockDim.x + threadIdx.x];

		likelihood[ly * size + lx] = likelihood[ly * size + lx] / sum;
	}
	else likelihood[ly * size + lx] = 0;
}

__global__ void run_EM_kernel(double *samples, double *weights, double *mu_mat, double *sig_mat, double *likelihood,
		double *marginals, int size, int dim, int num_gaus, double threshold, int max_iter)
{
	double change = 100;
	int iter = 0;
	double *tmp_data;
	int size_reduced = (THR_PER_BLOCK + size - 1) / THR_PER_BLOCK;

	dim3 grid_dim(1, 1, 1);
	dim3 block_dim(1, 1);
	int size_shared_mem;

	while (change > threshold && iter < max_iter)
	{
		/* E-step: Calculate normalized likelihood */

		block_dim.x = THR_PER_BLOCK;
		block_dim.y = 1;
		grid_dim.x = size_reduced;
		grid_dim.y = num_gaus;

		// compute weighted multivariate normal pdf for each slot
		w_mvnpdf_dim3<<<grid_dim, block_dim>>>(likelihood, samples, mu_mat, sig_mat, weights, size, num_gaus);
		hipDeviceSynchronize();

		// compute normalization
		block_dim.x = (num_gaus + THR_PER_BLOCK - 1) / num_gaus;
		block_dim.y = num_gaus;
		grid_dim.x = (block_dim.x + size - 1) / block_dim.x;
		grid_dim.y = 1;
		size_shared_mem = sizeof(double) * THR_PER_BLOCK;
		normalization<<<grid_dim, block_dim, size_shared_mem, 0>>>(likelihood, size, num_gaus);
		hipDeviceSynchronize();

		// M-step: Update weights, mus, sigmas
		block_dim.x = THR_PER_BLOCK;
		block_dim.y = 1;

		// update marginals
		grid_dim.x = size_reduced;
		grid_dim.y = num_gaus;

		tmp_data = (double *)malloc(sizeof(double) * grid_dim.x * grid_dim.y);
		marginal_reduction<<<grid_dim, block_dim>>>(likelihood, tmp_data, size, num_gaus);
		marginal_single_reduction<<<num_gaus, size_reduced>>>(tmp_data, marginals);
		free(tmp_data);
		hipDeviceSynchronize();

		// update mu
		grid_dim.x = size_reduced;
		grid_dim.y = dim;
		grid_dim.z = num_gaus;

		tmp_data = (double *)malloc(sizeof(double) * grid_dim.x * grid_dim.y * dim);
		mu_reduction<<<grid_dim, block_dim>>>(likelihood, samples, marginals, tmp_data, size, num_gaus, dim);

		grid_dim.x = dim;
		grid_dim.y = num_gaus;
		grid_dim.z = 1;
		single_reduction<<<grid_dim, size_reduced>>>(tmp_data, mu_mat);

		// update sigma
		grid_dim.x = size_reduced;
		grid_dim.y = dim * dim;
		grid_dim.z = num_gaus;

		tmp_data = (double *)malloc(sizeof(double) * grid_dim.x * grid_dim.y * grid_dim.z);
		sigma_reduction<<<grid_dim, block_dim>>>(likelihood, samples, marginals, mu_mat, tmp_data, size, num_gaus, dim);

		grid_dim.x = dim * dim;
		grid_dim.y = num_gaus;
		grid_dim.z = 1;
		single_reduction<<<grid_dim, size_reduced>>>(tmp_data, sig_mat);
		hipDeviceSynchronize();

		++iter;
	}
}

/**
 * Host function that copies the data and launches the work on GPU
 */

GaussianParam run_EM(double *samples, int s_size, int s_dim, int num_gaus, double threshold, int max_iter)
{
	int i, d, dim_squared = s_dim * s_dim;

	// Sizes for memory allocation
	int size_n_gaus = sizeof(double) * num_gaus;
	int size_n_samp = sizeof(double) * s_size;

	int size_sigma = sizeof(double) * dim_squared;
	int size_likelihood = sizeof(double) * num_gaus * s_size;
	int size_mu_mat = sizeof(double) * num_gaus * s_dim;
	int size_sig_mat = sizeof(double) * num_gaus * dim_squared;

	/* Allocate and initialize host (CPU) memory */
	double *likelihood = (double *) malloc(size_likelihood);
	double *likelihood_prev = (double *)malloc(size_likelihood);
	double *weights = (double *)malloc(size_n_gaus);
	double *mu_mat = (double *)malloc(size_mu_mat);
	double *sig_mat = (double *)malloc(size_sig_mat);

	GaussianParam *output = (GaussianParam *)malloc(sizeof(GaussianParam));

	/* Initialize Gaussian params and weights */
	time_t t;
	srand((unsigned)time(&t)); // seed random number generator

	double *inter_sigma, *trans_sigma, *init_sigma;
	for (i = 0; i < num_gaus; i++)
	{
		// Init mu randomly
		for (d = 0; d < s_dim; d++)
			mu_mat[i * s_dim + d] = ((double)rand() / RAND_MAX);

		// Init sigma randomly. To make sigma positive semi-definite, symmetric, sigma = s'*s
		inter_sigma = (double *)malloc(size_sigma);
		for (d = 0; d < dim_squared; d++)
			inter_sigma[d] = ((double)rand() / RAND_MAX);
		trans_sigma = transpose(inter_sigma, s_dim, s_dim);
		init_sigma = matrix_mult(trans_sigma, inter_sigma, s_dim, s_dim, s_dim);
		for (d = 0; d < dim_squared; ++d)
			sig_mat[i * dim_squared + d] = init_sigma[d];

		free(inter_sigma);
		free(trans_sigma);
		free(init_sigma);

		// Init weights uniformly
		weights[i] = 1 / (double) num_gaus;
	}

	/* Allocate device memory & copy data from host to device */
	double *d_samples, *d_weights, *d_likelihood, *d_mu_mat, *d_sig_mat, *d_marginals;

	// samples
	CUDA_CHECK_RETURN(hipMalloc( (void **) &d_samples, size_n_samp * s_dim));
	CUDA_CHECK_RETURN(hipMemcpy(d_samples, samples, size_n_samp * s_dim, hipMemcpyHostToDevice));

	// likelihood
	CUDA_CHECK_RETURN(hipMalloc( (void **) &d_likelihood, size_likelihood));
	CUDA_CHECK_RETURN(hipMemset(d_likelihood, 0, size_likelihood));

	// weights
	CUDA_CHECK_RETURN(hipMalloc( (void **) &d_weights, size_n_gaus));
	CUDA_CHECK_RETURN(hipMemcpy(d_weights, weights, size_n_gaus, hipMemcpyHostToDevice));

	// mu matrix
	CUDA_CHECK_RETURN(hipMalloc( (void **) &d_mu_mat, size_mu_mat));
	CUDA_CHECK_RETURN(hipMemcpy(d_mu_mat, mu_mat, size_mu_mat, hipMemcpyHostToDevice));

	// sigma matrix
	CUDA_CHECK_RETURN(hipMalloc( (void **) &d_sig_mat, size_sig_mat));
	CUDA_CHECK_RETURN(hipMemcpy(d_sig_mat, sig_mat, size_sig_mat, hipMemcpyHostToDevice));

	// marginals
	CUDA_CHECK_RETURN(hipMalloc( (void **) &d_marginals, size_n_gaus));
	CUDA_CHECK_RETURN(hipMemset(d_marginals, 0, size_n_gaus));

	run_EM_kernel<<<1, 1>>>(d_samples, d_weights, d_mu_mat, d_sig_mat, d_likelihood, d_marginals, s_size, s_dim, num_gaus, threshold, max_iter);


	CUDA_CHECK_RETURN(hipMemcpy(likelihood, d_likelihood, size_likelihood, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipFree(d_likelihood));

	print_mat(likelihood, num_gaus, s_size);


	/*
	while (change_L > threshold && iter < max_iter)
	{
		// E-step: Calculate normalized likelihood
		// TODO: develop a kernel
		for (j = 0; j < s_size; j++)
		{
			normalization = 0;
			for (k = 0; k < num_gaus; k++)
			{
				pdf = 1;
				normalization += weights[k] * pdf;
				// printf("sample idx = %d, guassian idx = %d, pdf = %e\n", j, k, pdf);
			}
			// printf("sample idx = %d, normalization = %e\n", j, normalization);
			if (normalization <= 0)
			{
				printf("sample idx = %d, INVALID normalization = %e\n", j, normalization);
				normalization = 1;
			}
			for (i = 0; i < num_gaus; i++)
				likelihood[i * s_size + j] = weights[i] * 1 / normalization;
		}

		//print_mat(likelihood, num_gaus, s_size);

		// M-step: update weights, means, covarience matrices
		for (i = 0; i < num_gaus; i++)
		{
			// reset the mu and sigma parameters to zero for updates
			// for (d = 0; d < s_dim; d++) output[i].mu[d] = 0;
			//for (d = 0; d < s_dim * s_dim; d++) output[i].sigma[d] = 0;
			dim_grid = (BLOCKSIZE_S + s_dim - 1) / BLOCKSIZE_S;
			dim_block = BLOCKSIZE_S;
			set_zeros_kernel<<<dim_grid, dim_block>>>(d_output[i].mu, s_dim, 1);
			set_zeros_kernel<<<dim_grid, dim_block>>>(d_output[i].sigma, s_dim, 1);

			dim_grid = (BLOCKSIZE + s_size - 1) / BLOCKSIZE;
			dim_block = BLOCKSIZE;

			// Compute marginal
			marginal = 0;
			// compute_marginal<<<dim_grid, dim_block>>>(&d_likelihood[i * s_size], &d_marginals[i], s_size);
			// CUDA_CHECK_RETURN(hipMemcpy(tmp_result, d_tmp_result, size_tmp, hipMemcpyDeviceToHost));
			// for (j = 0; j < dim_grid; j++) marginal += tmp_result[j];

			// Update weight
			weights[i] = marginal / s_size;

			// Update mean
			// TODO: develop kernel
			for (j = 0; j < s_size; j++)
				for (d = 0; d < s_dim; d++)
					output[i].mu[d] += likelihood[i * s_size + j] * samples[j];
			for (d = 0; d < s_dim; d++) output[i].mu[d] /= marginal;

			// Update covariance matrix
			// TODO: develop kernel
			for (j = 0; j < s_size; j++)
			{
				//mean_diff = matrix_subtr(samples[j], output[i].mu, s_dim, 1);
				//inter_sigma = matrix_mult(mean_diff, mean_diff, s_dim, 1, s_dim);
				//inter_sigma = matrix_scalar_mult(inter_sigma, inter_sigma, likelihood[i * s_size + j] / marginal, s_dim, s_dim);
				//matrix_add(output[i].sigma, output[i].sigma, inter_sigma, s_dim, s_dim);

				free(mean_diff);
				free(inter_sigma);
			}
		}

		// TODO: develop kernel
		change_L = 2; //eval_likelihood(likelihood_prev, likelihood, num_gaus, s_size);

		// Save likelihood matrix
		// TODO: develop kernel
		for (k = 0; k < s_size * num_gaus; k++)
			likelihood_prev[k] = likelihood[k];
		iter++;
		printf("ITER = %d\nchange_L = %e", iter, change_L);
	}
	*/

	/* Copy data from device to host & free device memory */
	CUDA_CHECK_RETURN(hipMemcpy(weights, d_weights, size_n_gaus, hipMemcpyDeviceToHost)); // weights
	CUDA_CHECK_RETURN(hipFree(d_weights));

	CUDA_CHECK_RETURN(hipMemcpy(mu_mat, d_mu_mat, size_mu_mat, hipMemcpyDeviceToHost)); //
	CUDA_CHECK_RETURN(hipFree(d_mu_mat));

	CUDA_CHECK_RETURN(hipMemcpy(sig_mat, d_sig_mat, size_sig_mat, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipFree(d_sig_mat));

	CUDA_CHECK_RETURN(hipMemcpy(likelihood, d_likelihood, size_likelihood, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipFree(d_likelihood));

	output->mu = mu_mat;
	output->sigma = sig_mat;

	return output[0];
}

int main()
{
	int size = sizeof(double) * 3;
	double h_x[3] = {0.3188, -1.3077, -0.4336};
	double h_mu[3] = {0, 0, 0};
	double h_sig[9] = {1, 0, 0, 0, 1, 0, 0, 0, 1};
	double pdf;

	double *d_x, *d_mu, *d_sig, *d_pdf;
	hipMalloc((void **)&d_x, size);
	hipMalloc((void **)&d_mu, size);
	hipMalloc((void **)&d_sig, size * 3);
	hipMalloc((void **)&d_pdf, sizeof(double));

	hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
	hipMemcpy(d_mu, h_mu, size, hipMemcpyHostToDevice);
	hipMemcpy(d_sig, h_sig, size * 3, hipMemcpyHostToDevice);

	mvnpdf_dim3<<<1, 1>>>(d_pdf, d_x, d_mu, d_sig, 1, 1);

	hipMemcpy(&pdf, d_pdf, sizeof(double), hipMemcpyDeviceToHost);

	for (int i = 0; i < 10; ++i)
		std::cout << "pdf = " << pdf << std::endl;


	return 0;
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned " << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}

