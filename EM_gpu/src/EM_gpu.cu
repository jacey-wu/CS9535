#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : EM_gpu.cu
 Author      : Jiaxiao Wu
 Version     :
 Copyright   : 
 Description : CUDA expectation-maximization algorithm
 ============================================================================
 */

#include <iostream>
#include <numeric>
#include <cstdlib>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cfloat>

#include "utillib/helper_string.h"
#include "utillib/hip/hip_runtime_api.h"

#include "EM_gpu.h"
extern "C" {
#include "matrix.h"
}

static void CheckCudaErrorAux (const char *, unsigned, const char *, hipError_t);

#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)
#define WARPSIZE 32
#define THR_PER_BLOCK_SM 128
#define THR_PER_BLOCK 256

#define MATH_PI 3.14159265358979323846   // pi
#define MEAN_PRIOR 5;
#define VAR_PRIOR 2;


/**
 * CUDA kernels
 */

__global__ void marginal_single_reduction(double *g_idata, double *g_odata)
{
	extern __shared__ double sdata[];
	int tid = threadIdx.x;

	// Read data into shared memory
	sdata[tid] = g_idata[blockIdx.x * blockDim.x + tid];
	__syncthreads();

	// Reduction in shared memory
	for (int s = blockDim.x / 2; s > WARPSIZE; s>>=1)    // Reversed loop
	{
		if (tid < s)                 // Non-divergent branch
			sdata[tid] += sdata[tid + s]; // Sequential addressing
		__syncthreads();
	}

	if (tid < WARPSIZE)
	{
		if (blockDim.x >= 64) sdata[tid] += sdata[tid + 32];
		if (blockDim.x >= 32) sdata[tid] += sdata[tid + 16];
		if (blockDim.x >= 16) sdata[tid] += sdata[tid + 8];
		if (blockDim.x >= 8) sdata[tid] += sdata[tid + 4];
		if (blockDim.x >= 4) sdata[tid] += sdata[tid + 2];
		if (blockDim.x >= 2) sdata[tid] += sdata[tid + 1];
	}

	// Write the sum of this block to device memory
	if (tid == 0) g_odata[blockIdx.x] = sdata[0];
}

__global__ void marginal_reduction(double *likelihood, double *tmp_odata, int size, int num_gaus)
{
	int tid = threadIdx.x;
	int lx = blockDim.x * blockIdx.x + threadIdx.x;
	int ly = blockIdx.y;

	if (lx < size && ly < num_gaus)
	{
		// Read data into shared memory
		extern __shared__ double sdata[];
		sdata[tid] = likelihood[ly * size + lx];
		__syncthreads();

		// Reduction in shared memory
		for (int s = blockDim.x / 2; s > WARPSIZE; s>>=1)    // Reversed loop
		{
			if (tid < s)                 // Non-divergent branch
				sdata[tid] += sdata[tid + s]; // Sequential addressing
			__syncthreads();
		}

		if (tid < WARPSIZE)
		{
			if (blockDim.x >= 64) sdata[tid] += sdata[tid + 32];
			if (blockDim.x >= 32) sdata[tid] += sdata[tid + 16];
			if (blockDim.x >= 16) sdata[tid] += sdata[tid + 8];
			if (blockDim.x >= 8) sdata[tid] += sdata[tid + 4];
			if (blockDim.x >= 4) sdata[tid] += sdata[tid + 2];
			if (blockDim.x >= 2) sdata[tid] += sdata[tid + 1];
		}

		// Write the sum of this block to device memory
		if (tid == 0) tmp_odata[blockIdx.y * gridDim.x + blockIdx.x] = sdata[0];

	}
}

__global__ void weight_update(double *weights, double *marginals, int num_gaus, int size)
{
	int tid = threadIdx.x;
	if (tid < num_gaus)
	{
		double sum = 0;
		for (int i = 0; i < num_gaus; ++i) sum += marginals[i];

		if (marginals[tid] == 0)
			weights[tid] = DBL_MIN;
		else
			weights[tid] = marginals[tid] / (double) sum;
	}
}

__global__ void weight_update_orig(double *weights, double *marginals, int num_gaus, int size)
{
	int tid = threadIdx.x;
	if (tid < num_gaus)
	{
		if (marginals[tid] == 0)
			weights[tid] = DBL_MIN;
		else
			weights[tid] = marginals[tid] / (double) size;
	}
}

__global__ void mu_reduction(double *likelihood, double *samples, double *marg, double *tmp_odata, int size, int num_gaus, int dim)
{
	int tid = threadIdx.x;
	int blockId = (gridDim.x * gridDim.y) * blockIdx.z + gridDim.x * blockIdx.y + blockIdx.x;

	int samp_idx = blockIdx.x * blockDim.x + threadIdx.x;
	int dim_idx = blockIdx.y;
	int gaus_idx = blockIdx.z;

	if (samp_idx < size && dim_idx < dim && gaus_idx < num_gaus)
	{
		// Read data into shared memory
		extern __shared__ double sdata[];
		sdata[tid] = likelihood[gaus_idx * size + samp_idx] * samples[samp_idx * dim + dim_idx];
		__syncthreads();

		// Reduction in shared memory
		for (int s = blockDim.x / 2; s > WARPSIZE; s>>=1)    // Reversed loop
		{
			if (tid < s)                      // Non-divergent branch
				sdata[tid] += sdata[tid + s]; // Sequential addressing
			__syncthreads();
		}

		if (tid < WARPSIZE)
		{
			if (blockDim.x >= 64) sdata[tid] += sdata[tid + 32];
			if (blockDim.x >= 32) sdata[tid] += sdata[tid + 16];
			if (blockDim.x >= 16) sdata[tid] += sdata[tid + 8];
			if (blockDim.x >= 8) sdata[tid] += sdata[tid + 4];
			if (blockDim.x >= 4) sdata[tid] += sdata[tid + 2];
			if (blockDim.x >= 2) sdata[tid] += sdata[tid + 1];
		}

		// Write the sum of this block to device memory
		if (tid == 0) tmp_odata[blockId] = sdata[0] / (double) marg[gaus_idx];
	}
}

__global__ void sigma_reduction(double *likelihood, double *samples, double *marg, double *mu, double *odata,
		int size, int num_gaus, int dim)
{
	int tid = threadIdx.x;
	int blockId = (gridDim.x * gridDim.y) * blockIdx.z + gridDim.x * blockIdx.y + blockIdx.x;

	int samp_idx = blockIdx.x * blockDim.x + threadIdx.x;
	int gaus_idx = blockIdx.z;

	int sigy = blockIdx.y / dim;
	int sigx = blockIdx.y - dim * sigy;

	if (samp_idx < size && gaus_idx < num_gaus && sigx < dim && sigy < dim)
	{
		// Read data into shared memory
		extern __shared__ double sdata[];

		sdata[tid] = likelihood[gaus_idx * size + samp_idx]
		                     * (samples[samp_idx * dim + sigx] - mu[gaus_idx * dim + sigx]) // non-coalesced access
		                     * (samples[samp_idx * dim + sigy] - mu[gaus_idx * dim + sigy]);
		__syncthreads();

		// Reduction in shared memory
		for (int s = blockDim.x / 2; s > WARPSIZE; s>>=1)    // Reversed loop
		{
			if (tid < s)                 // Non-divergent branch
				sdata[tid] += sdata[tid + s]; // Sequential addressing
			__syncthreads();
		}

		if (tid < WARPSIZE)
		{
			if (blockDim.x >= 64) sdata[tid] += sdata[tid + 32];
			if (blockDim.x >= 32) sdata[tid] += sdata[tid + 16];
			if (blockDim.x >= 16) sdata[tid] += sdata[tid + 8];
			if (blockDim.x >= 8)  sdata[tid] += sdata[tid + 4];
			if (blockDim.x >= 4)  sdata[tid] += sdata[tid + 2];
			if (blockDim.x >= 2)  sdata[tid] += sdata[tid + 1];
		}

		// Write the sum of this block to device memory
		if (tid == 0) odata[blockId] = sdata[0] / (double) marg[gaus_idx];
	}
}

__global__ void single_reduction(double *idata, double *omat)
{
	extern __shared__ double sdata[];
	int tid = threadIdx.x;
	int gid = (blockIdx.y * gridDim.x + blockIdx.x) * blockDim.x + threadIdx.x;

	// Read data into shared memory
	sdata[tid] = idata[gid];
	__syncthreads();

	// Reduction in shared memory
	for (int s = blockDim.x / 2; s > WARPSIZE; s>>=1)    // Reversed loop
	{
		if (tid < s)                 // Non-divergent branch
			sdata[tid] += sdata[tid + s]; // Sequential addressing
		__syncthreads();
	}

	if (tid < WARPSIZE)
	{
		if (blockDim.x >= 64) sdata[tid] += sdata[tid + 32];
		if (blockDim.x >= 32) sdata[tid] += sdata[tid + 16];
		if (blockDim.x >= 16) sdata[tid] += sdata[tid + 8];
		if (blockDim.x >= 8) sdata[tid] += sdata[tid + 4];
		if (blockDim.x >= 4) sdata[tid] += sdata[tid + 2];
		if (blockDim.x >= 2) sdata[tid] += sdata[tid + 1];
	}

	// Write the sum of this block to device memory
	if (tid == 0) omat[blockIdx.y * gridDim.x + blockIdx.x] = sdata[0];
}

__global__ void mvnpdf_dim3(double *likelihood, double *samples, double *mu_mat, double *sig_mat, int size, int num_gaus)
{
	const int dim = 3;
	int lx = blockDim.x * blockIdx.x + threadIdx.x;
	int ly = blockIdx.y;
	if (lx < size && ly < num_gaus)
	{
		int i;
		__shared__ double x[dim];
		__shared__ double mu[dim];
		__shared__ double sigma[dim * dim];
		__shared__ double inv_sig[dim * dim];
		__shared__ double d[dim];

		// Read data from global to shared mem
		for (i = 0; i < dim; ++i)
		{
			x[i] = samples[lx * dim + i];
			mu[i] = mu_mat[ly * dim + i];
		}
		for (i = 0; i < dim * dim; ++i) sigma[i] = sig_mat[ly * dim * dim + i];


		// determinant and inverse of sigma
		double det = sigma[0] * (sigma[4] * sigma[8] - sigma[5] * sigma[7])
				- sigma[1] * (sigma[3] * sigma[8] - sigma[5] * sigma[6])
				+ sigma[2] * (sigma[3] * sigma[7] - sigma[4] * sigma[6]);

		inv_sig[0] = (sigma[4] * sigma[8] - sigma[5] * sigma[7]) / (double) det;
		inv_sig[1] = (sigma[2] * sigma[7] - sigma[1] * sigma[8]) / (double) det;
		inv_sig[2] = (sigma[1] * sigma[5] - sigma[2] * sigma[4]) / (double) det;
		inv_sig[3] = (sigma[5] * sigma[6] - sigma[3] * sigma[8]) / (double) det;
		inv_sig[4] = (sigma[0] * sigma[8] - sigma[2] * sigma[6]) / (double) det;
		inv_sig[5] = (sigma[2] * sigma[3] - sigma[0] * sigma[5]) / (double) det;
		inv_sig[6] = (sigma[3] * sigma[7] - sigma[4] * sigma[6]) / (double) det;
		inv_sig[7] = (sigma[1] * sigma[6] - sigma[0] * sigma[7]) / (double) det;
		inv_sig[8] = (sigma[0] * sigma[4] - sigma[1] * sigma[3]) / (double) det;

		// diff = x[i] - mu[i]
		for (i = 0; i < dim; ++i) d[i] = x[i] - mu[i];

		// expon = -1/2 * (x - mu)' * inv_sig * (x - mu)
		double expon = ((d[0]*inv_sig[0] + d[1]*inv_sig[3] + d[2]*inv_sig[6]) * d[0]
                      + (d[0]*inv_sig[1] + d[1]*inv_sig[4] + d[2]*inv_sig[7]) * d[1]
                      + (d[0]*inv_sig[2] + d[1]*inv_sig[5] + d[2]*inv_sig[8]) * d[2]) / (double) -2;

		// denom = sqrt((2pi)^dim * det(sigma))
		double denom = sqrt(pow(2 * MATH_PI, dim) * det);

		likelihood[ly * size + lx] = exp(expon) / (double) denom;
	}
	else likelihood[ly * size + lx] = 0;
}

__global__ void w_mvnpdf_dim3(double *likelihood, double *samples, double *mu_mat, double *sig_mat,
		double *weights, int size, int num_gaus)
{
	int lx = blockDim.x * blockIdx.x + threadIdx.x;
	int ly = blockIdx.y;
	if (lx < size && ly < num_gaus)
	{
		int i, dim = 3;
		int os_samp = dim * threadIdx.x;
		int os_sig = dim * dim * threadIdx.x;
		int thr_per_block = blockDim.x * blockDim.y;

		extern __shared__ double sdata[];
		double *x = (double *) &sdata[0];
		double *mu = (double *) &x[dim * thr_per_block];
		double *diff = (double *) &mu[dim * thr_per_block];
		double *sigma = (double *) &diff[dim * thr_per_block];
		double *inv_sig = (double *) &sigma[dim * dim * thr_per_block];

		double weight = weights[ly];

		// Read data from global to shared mem
		for (i = 0; i < dim; ++i)
		{
			x[os_samp + i] = samples[lx * dim + i];
			// x[i] = 12;
			mu[os_samp + i] = mu_mat[ly * dim + i];
		}
		for (i = 0; i < dim * dim; ++i) sigma[os_sig + i] = sig_mat[ly * dim * dim + i];

		// determinant and inverse of sigma
		double det = sigma[os_sig + 0] * (sigma[os_sig + 4] * sigma[os_sig + 8] - sigma[os_sig + 5] * sigma[os_sig + 7])
				   - sigma[os_sig + 1] * (sigma[os_sig + 3] * sigma[os_sig + 8] - sigma[os_sig + 5] * sigma[os_sig + 6])
				   + sigma[os_sig + 2] * (sigma[os_sig + 3] * sigma[os_sig + 7] - sigma[os_sig + 4] * sigma[os_sig + 6]);

		inv_sig[os_sig + 0] = (sigma[os_sig + 4] * sigma[os_sig + 8] - sigma[os_sig + 5] * sigma[os_sig + 7]) / (double) det;
		inv_sig[os_sig + 1] = (sigma[os_sig + 2] * sigma[os_sig + 7] - sigma[os_sig + 1] * sigma[os_sig + 8]) / (double) det;
		inv_sig[os_sig + 2] = (sigma[os_sig + 1] * sigma[os_sig + 5] - sigma[os_sig + 2] * sigma[os_sig + 4]) / (double) det;
		inv_sig[os_sig + 3] = (sigma[os_sig + 5] * sigma[os_sig + 6] - sigma[os_sig + 3] * sigma[os_sig + 8]) / (double) det;
		inv_sig[os_sig + 4] = (sigma[os_sig + 0] * sigma[os_sig + 8] - sigma[os_sig + 2] * sigma[os_sig + 6]) / (double) det;
		inv_sig[os_sig + 5] = (sigma[os_sig + 2] * sigma[os_sig + 3] - sigma[os_sig + 0] * sigma[os_sig + 5]) / (double) det;
		inv_sig[os_sig + 6] = (sigma[os_sig + 3] * sigma[os_sig + 7] - sigma[os_sig + 4] * sigma[os_sig + 6]) / (double) det;
		inv_sig[os_sig + 7] = (sigma[os_sig + 1] * sigma[os_sig + 6] - sigma[os_sig + 0] * sigma[os_sig + 7]) / (double) det;
		inv_sig[os_sig + 8] = (sigma[os_sig + 0] * sigma[os_sig + 4] - sigma[os_sig + 1] * sigma[os_sig + 3]) / (double) det;

		// diff = x[i] - mu[i]
		for (i = 0; i < dim; ++i) diff[os_samp + i] = x[os_samp + i] - mu[os_samp + i];

		// expon = -1/2 * (x - mu)' * inv_sig * (x - mu)
		double expon = (double) (-1) / 2
				  *((diff[os_samp + 0]*inv_sig[os_sig + 0] + diff[os_samp + 1]*inv_sig[os_sig + 3] + diff[os_samp + 2]*inv_sig[os_sig + 6]) * diff[os_samp + 0]
                  + (diff[os_samp + 0]*inv_sig[os_sig + 1] + diff[os_samp + 1]*inv_sig[os_sig + 4] + diff[os_samp + 2]*inv_sig[os_sig + 7]) * diff[os_samp + 1]
                  + (diff[os_samp + 0]*inv_sig[os_sig + 2] + diff[os_samp + 1]*inv_sig[os_sig + 5] + diff[os_samp + 2]*inv_sig[os_sig + 8]) * diff[os_samp + 2]);

		// denom = sqrt((2pi)^dim * det(sigma))
		double denom = sqrt(pow(2 * MATH_PI, dim) * fabs(det));

		double value = weight * exp(expon) / (double) denom;

		if (det == 0) 							// Sig matrix not invertable
			likelihood[ly * size + lx] = 0;
		else if (isinf(value)) 						// Large enough to exceed precision
			likelihood[ly * size + lx] = DBL_MAX;
		else
			likelihood[ly * size + lx] = value;
	}
	else
		likelihood[ly * size + lx] = 0;
}

__global__ void normalization(double *likelihood, int size, int num_gaus)
{
	int tid = threadIdx.y * blockDim.x + threadIdx.x;
	int lx = blockDim.x * blockIdx.x + threadIdx.x;
	int ly = threadIdx.y;

	if (lx < size && ly < num_gaus)
	{
		int i;
		double sum = 0;
		extern __shared__ double sdata[];

		// read data from global to shared mem
		sdata[tid] = likelihood[ly * size + lx];
		__syncthreads();

		// sum up likelihood values for the same sample
		for (i = 0; i < num_gaus; ++i)
			sum = sum + sdata[i * blockDim.x + threadIdx.x];

		if (sum == 0) sum = 1;
//			likelihood[ly * size + lx] = 1 / (double) num_gaus;
//		else if (likelihood[ly * size + lx] == 0)
//			likelihood[ly * size + lx] = DBL_MIN;
//		else
			likelihood[ly * size + lx] = sdata[tid] / (double) sum;
	}
	else likelihood[ly * size + lx] = 0;
}

/**
 * Host function that copies the data and launches the work on GPU
 */

GaussianParam run_EM(double *samples, int s_size, int s_dim, int num_gaus, double threshold, int max_iter, bool use_timer)
{
	int i, d, dim_squared = s_dim * s_dim;

	// Sizes for memory allocation
	int size_n_gaus = sizeof(double) * num_gaus;
	int size_n_samp = sizeof(double) * s_size;

	int size_sigma = sizeof(double) * dim_squared;
	int size_likelihood = sizeof(double) * num_gaus * s_size;
	int size_mu_mat = sizeof(double) * num_gaus * s_dim;
	int size_sig_mat = sizeof(double) * num_gaus * dim_squared;

	/* Allocate and initialize host (CPU) memory */
	double *likelihood = (double *) malloc(size_likelihood);
	double *likelihood_prev = (double *)malloc(size_likelihood);
	double *weights = (double *)malloc(size_n_gaus);
	double *mu_mat = (double *)malloc(size_mu_mat);
	double *sig_mat = (double *)malloc(size_sig_mat);

	// debug vars
	double *marginals = (double *)malloc(size_n_gaus);
	// double *tmp_data;
	double *d_tmp_data;

	GaussianParam *output = (GaussianParam *)malloc(sizeof(GaussianParam));

	/* Initialize Gaussian params and weights */
	time_t t;
	srand((unsigned)time(&t)); // seed random number generator

	double *inter_sigma, *trans_sigma, *init_sigma;

	for (i = 0; i < num_gaus; i++)
	{
		// Init mu randomly
		for (d = 0; d < s_dim; d++)
			mu_mat[i * s_dim + d] = ((double)rand() / RAND_MAX) * MEAN_PRIOR;

		// Init sigma randomly. To make sigma positive semi-definite, symmetric, sigma = s'*s
		inter_sigma = (double *)malloc(size_sigma);
		for (d = 0; d < dim_squared; d++)
			inter_sigma[d] = ((double)rand() / RAND_MAX) * VAR_PRIOR;
		trans_sigma = transpose(inter_sigma, s_dim, s_dim);
		init_sigma = matrix_mult(trans_sigma, inter_sigma, s_dim, s_dim, s_dim);
		for (d = 0; d < dim_squared; ++d)
			sig_mat[i * dim_squared + d] = init_sigma[d];

		free(inter_sigma);
		free(trans_sigma);
		free(init_sigma);

		// Init weights uniformly
		weights[i] = 1 / (double) num_gaus;
	}

	/* Allocate device memory & copy data from host to device */
	double *d_samples, *d_weights, *d_likelihood, *d_mu_mat, *d_sig_mat, *d_marginals;

	// samples
	CUDA_CHECK_RETURN(hipMalloc( (void **) &d_samples, size_n_samp * s_dim));
	CUDA_CHECK_RETURN(hipMemcpy(d_samples, samples, size_n_samp * s_dim, hipMemcpyHostToDevice));

	// likelihood
	CUDA_CHECK_RETURN(hipMalloc( (void **) &d_likelihood, size_likelihood));
	CUDA_CHECK_RETURN(hipMemset(d_likelihood, 0, size_likelihood));

	// weights
	CUDA_CHECK_RETURN(hipMalloc( (void **) &d_weights, size_n_gaus));
	CUDA_CHECK_RETURN(hipMemcpy(d_weights, weights, size_n_gaus, hipMemcpyHostToDevice));

	// mu matrix
	CUDA_CHECK_RETURN(hipMalloc( (void **) &d_mu_mat, size_mu_mat));
	CUDA_CHECK_RETURN(hipMemcpy(d_mu_mat, mu_mat, size_mu_mat, hipMemcpyHostToDevice));

	// sigma matrix
	CUDA_CHECK_RETURN(hipMalloc( (void **) &d_sig_mat, size_sig_mat));
	CUDA_CHECK_RETURN(hipMemcpy(d_sig_mat, sig_mat, size_sig_mat, hipMemcpyHostToDevice));

	// marginals
	CUDA_CHECK_RETURN(hipMalloc( (void **) &d_marginals, size_n_gaus));
	CUDA_CHECK_RETURN(hipMemset(d_marginals, 0, size_n_gaus));

	/* Use CUDA event to time kernel */
    hipEvent_t start, stop;
    checkCudaErrors(hipEventCreate(&start));
    checkCudaErrors(hipEventCreate(&stop));
    float msec;

    // Kernel-wise timer
    hipEvent_t ker_start, ker_stop;
    float ker_msec;
    EmTimer timer;
    if (use_timer)
    {	// Set time accumulators to be 0
        checkCudaErrors(hipEventCreate(&ker_start));
        checkCudaErrors(hipEventCreate(&ker_stop));
		timer.mvnpdf = 0;
		timer.normalization = 0;
		timer.marginal_red = 0;
		timer.marginal_red_sig = 0;
		timer.weight_update = 0;
		timer.mu_red = 0;
		timer.mu_red_sig = 0;
		timer.sigma_red = 0;
		timer.sigma_red_sig = 0;
    }

	/* ---------------- Interative update ------------------ */
	double change = 100;
	int iter = 0;
	int size_reduced = (THR_PER_BLOCK + s_size - 1) / THR_PER_BLOCK;
	int size_reduced_sm = (THR_PER_BLOCK_SM + s_size - 1) / THR_PER_BLOCK_SM;

	dim3 grid_dim(1, 1, 1);
	dim3 block_dim(1, 1);
	int size_shared_mem;

	// Start timer
	checkCudaErrors(hipEventRecord(start, 0));

	while (iter < max_iter)
	{
		//*******************************************//
		//* E-step: Calculate normalized likelihood *//
		//*******************************************//

		/* compute weighted multivariate normal pdf */
		block_dim.x = THR_PER_BLOCK_SM; // Use less threads per block due to the limitation of shared memory size
		block_dim.y = 1;
		grid_dim.x = size_reduced_sm;
		grid_dim.y = num_gaus;
		size_shared_mem = sizeof(double) * THR_PER_BLOCK_SM * (3 * s_dim + 2 * s_dim * s_dim);

		// Start kernel timer
		if (use_timer) checkCudaErrors(hipEventRecord(ker_start, 0));
		w_mvnpdf_dim3<<<grid_dim, block_dim, size_shared_mem>>>(d_likelihood, d_samples, d_mu_mat, d_sig_mat, d_weights, s_size, num_gaus);

		// Check kernel timer
		if (use_timer)
		{
			checkCudaErrors(hipEventRecord(ker_stop, 0));
			checkCudaErrors(hipEventSynchronize(ker_stop));
			checkCudaErrors(hipEventElapsedTime(&ker_msec, ker_start, ker_stop));
			timer.mvnpdf += ker_msec;
		}
		else hipDeviceSynchronize();

		/* -- check likelihood
		CUDA_CHECK_RETURN(hipMemcpy(likelihood, d_likelihood, size_likelihood, hipMemcpyDeviceToHost));
		hipDeviceSynchronize();
		std::cout << "Likelihood at iter = " << iter << std::endl; // PRINT
		print_mat(likelihood, num_gaus, s_size);
		std::cout << std::endl;
		*/

		/* compute normalization */
		block_dim.x = (num_gaus + THR_PER_BLOCK - 1) / num_gaus;
		block_dim.y = num_gaus;
		grid_dim.x = (block_dim.x + s_size - 1) / block_dim.x;
		grid_dim.y = 1;
		size_shared_mem = sizeof(double) * block_dim.x * block_dim.y;

		// Start kernel timer
		if (use_timer) checkCudaErrors(hipEventRecord(ker_start, 0));
		normalization<<<grid_dim, block_dim, size_shared_mem, 0>>>(d_likelihood, s_size, num_gaus);

		// Check kernel timer
		if (use_timer)
		{
			checkCudaErrors(hipEventRecord(ker_stop, 0));
			checkCudaErrors(hipEventSynchronize(ker_stop));
			checkCudaErrors(hipEventElapsedTime(&ker_msec, ker_start, ker_stop));
			timer.normalization += ker_msec;
		}
		else hipDeviceSynchronize();

		/* -- check normalization
		CUDA_CHECK_RETURN(hipMemcpy(likelihood, d_likelihood, size_likelihood, hipMemcpyDeviceToHost));
		hipDeviceSynchronize();
		std::cout << "Normalized Likelihood at iter = " << iter << std::endl; // PRINT
		print_mat(likelihood, num_gaus, s_size);
		std::cout << std::endl;
		*/

		//*******************************************//
		// *   M-step: Update weights, mus, sigmas  *//
		//*******************************************//
		block_dim.x = THR_PER_BLOCK;
		block_dim.y = 1;
		size_shared_mem = sizeof(double) * THR_PER_BLOCK;

		/* update marginals */
		grid_dim.x = size_reduced;
		grid_dim.y = num_gaus;

		// tmp_data = (double *) malloc(sizeof(double) * grid_dim.x * grid_dim.y);
		CUDA_CHECK_RETURN(hipMalloc( (void **) &d_tmp_data, sizeof(double) * grid_dim.x * grid_dim.y));

		// Start kernel timer
		if (use_timer) checkCudaErrors(hipEventRecord(ker_start, 0));
		marginal_reduction<<<grid_dim, block_dim, size_shared_mem>>>(d_likelihood, d_tmp_data, s_size, num_gaus);

		// Check kernel timer
		if (use_timer)
		{
			checkCudaErrors(hipEventRecord(ker_stop, 0));
			checkCudaErrors(hipEventSynchronize(ker_stop));
			checkCudaErrors(hipEventElapsedTime(&ker_msec, ker_start, ker_stop));
			timer.marginal_red += ker_msec;
		}
		else hipDeviceSynchronize();

		/* -- check tmp data of marginals
		CUDA_CHECK_RETURN(hipMemcpy(tmp_data, d_tmp_data, sizeof(double) * grid_dim.x * grid_dim.y, hipMemcpyDeviceToHost));
		hipDeviceSynchronize();
		std::cout << "tmp marginals at iter = " << iter << std::endl; // PRINT
		print_mat(tmp_data, grid_dim.y, grid_dim.x);
		std::cout << std::endl;
		*/

		// Start kernel timer
		if (use_timer) checkCudaErrors(hipEventRecord(ker_start, 0));
		marginal_single_reduction<<<num_gaus, size_reduced, sizeof(double) * size_reduced>>>(d_tmp_data, d_marginals);

		// Check kernel timer
		if (use_timer)
		{
			checkCudaErrors(hipEventRecord(ker_stop, 0));
			checkCudaErrors(hipEventSynchronize(ker_stop));
			checkCudaErrors(hipEventElapsedTime(&ker_msec, ker_start, ker_stop));
			timer.marginal_red_sig += ker_msec;
		}
		else hipDeviceSynchronize();
		CUDA_CHECK_RETURN(hipFree(d_tmp_data));

		/* -- check marginals
		CUDA_CHECK_RETURN(hipMemcpy(marginals, d_marginals, size_n_gaus, hipMemcpyDeviceToHost));
		hipDeviceSynchronize();
		std::cout << "Marginals at iter = " << iter << std::endl; // PRINT
		print_mat(marginals, 1, num_gaus);
		std::cout << std::endl;
		*/

		/* Update weights */

		// Start kernel timer
		if (use_timer) checkCudaErrors(hipEventRecord(ker_start, 0));
		weight_update<<<1, num_gaus>>>(d_weights, d_marginals, num_gaus, s_size);

		// Check kernel timer
		if (use_timer)
		{
			checkCudaErrors(hipEventRecord(ker_stop, 0));
			checkCudaErrors(hipEventSynchronize(ker_stop));
			checkCudaErrors(hipEventElapsedTime(&ker_msec, ker_start, ker_stop));
			timer.weight_update += ker_msec;
		}
		// else hipDeviceSynchronize();

		/* -- check weights
		CUDA_CHECK_RETURN(hipMemcpy(weights, d_weights, size_n_gaus, hipMemcpyDeviceToHost));
		hipDeviceSynchronize();
		std::cout << "Weights at iter = " << iter << std::endl; // PRINT
		print_mat(weights, 1, num_gaus);
		std::cout << std::endl;
		*/

		/* Update mu */
		grid_dim.x = size_reduced;
		grid_dim.y = s_dim;
		grid_dim.z = num_gaus;
		CUDA_CHECK_RETURN(hipMalloc( (void **) &d_tmp_data, sizeof(double) * grid_dim.x * grid_dim.y * s_dim));

		// Start kernel timer
		if (use_timer) checkCudaErrors(hipEventRecord(ker_start, 0));
		mu_reduction<<<grid_dim, block_dim, size_shared_mem>>>(d_likelihood, d_samples, d_marginals, d_tmp_data, s_size, num_gaus, s_dim);

		// Check kernel timer
		if (use_timer)
		{
			checkCudaErrors(hipEventRecord(ker_stop, 0));
			checkCudaErrors(hipEventSynchronize(ker_stop));
			checkCudaErrors(hipEventElapsedTime(&ker_msec, ker_start, ker_stop));
			timer.mu_red += ker_msec;
		}
		else hipDeviceSynchronize();

		grid_dim.x = s_dim;
		grid_dim.y = num_gaus;
		grid_dim.z = 1;

		// Start kernel timer
		if (use_timer) checkCudaErrors(hipEventRecord(ker_start, 0));
		single_reduction<<<grid_dim, size_reduced, size_shared_mem>>>(d_tmp_data, d_mu_mat);

		// Check kernel timer
		if (use_timer)
		{
			checkCudaErrors(hipEventRecord(ker_stop, 0));
			checkCudaErrors(hipEventSynchronize(ker_stop));
			checkCudaErrors(hipEventElapsedTime(&ker_msec, ker_start, ker_stop));
			timer.mu_red_sig += ker_msec;
		}
		else hipDeviceSynchronize();
		hipFree(d_tmp_data);

		/* -- check mu_mat
		CUDA_CHECK_RETURN(hipMemcpy(mu_mat, d_mu_mat, size_mu_mat, hipMemcpyDeviceToHost));
		hipDeviceSynchronize();
		std::cout << "Mu_mat at iter = " << iter << std::endl; // PRINT
		print_mat(mu_mat, num_gaus, s_dim);
		std::cout << std::endl;
		*/

		/* Update sigma */
		grid_dim.x = size_reduced;
		grid_dim.y = s_dim * s_dim;
		grid_dim.z = num_gaus;

		CUDA_CHECK_RETURN(hipMalloc( (void **) &d_tmp_data, sizeof(double) * grid_dim.x * grid_dim.y * grid_dim.z));

		// Start kernel timer
		if (use_timer) checkCudaErrors(hipEventRecord(ker_start, 0));
		sigma_reduction<<<grid_dim, block_dim, size_shared_mem>>>(d_likelihood, d_samples, d_marginals, d_mu_mat, d_tmp_data, s_size, num_gaus, s_dim);

		// Check kernel timer
		if (use_timer)
		{
			checkCudaErrors(hipEventRecord(ker_stop, 0));
			checkCudaErrors(hipEventSynchronize(ker_stop));
			checkCudaErrors(hipEventElapsedTime(&ker_msec, ker_start, ker_stop));
			timer.sigma_red += ker_msec;
		}
		else hipDeviceSynchronize();

		grid_dim.x = s_dim * s_dim;
		grid_dim.y = num_gaus;
		grid_dim.z = 1;

		// Start kernel timer
		if (use_timer) checkCudaErrors(hipEventRecord(ker_start, 0));
		single_reduction<<<grid_dim, size_reduced, size_shared_mem>>>(d_tmp_data, d_sig_mat);

		// Check kernel timer
		if (use_timer)
		{
			checkCudaErrors(hipEventRecord(ker_stop, 0));
			checkCudaErrors(hipEventSynchronize(ker_stop));
			checkCudaErrors(hipEventElapsedTime(&ker_msec, ker_start, ker_stop));
			timer.sigma_red_sig += ker_msec;
		}
		else hipDeviceSynchronize();

		hipFree(d_tmp_data);

		/* -- check sig_mat
		CUDA_CHECK_RETURN(hipMemcpy(sig_mat, d_sig_mat, size_sig_mat, hipMemcpyDeviceToHost));
		hipDeviceSynchronize();
		std::cout << "Sig_mat at iter = " << iter << std::endl; // PRINT
		print_mat(sig_mat, num_gaus, dim_squared);
		std::cout << std::endl;
		*/

		++iter;
	}
	/* --------------------------------------- */

	// Stop timer
	checkCudaErrors(hipEventRecord(stop, 0));
	checkCudaErrors(hipEventSynchronize(stop));
	checkCudaErrors(hipEventElapsedTime(&msec, start, stop));

	/* Print experiment summary */
	printf("All EM kernels: TotTime = %.4f ms; NumIter = %d; TimePerIter = %.4f ms \n", msec, max_iter, msec / max_iter);

	if (use_timer)
	{
		printf("Mvnpdf kernel:                    TotTime = %.4f ms; NumIter = %d; TimePerIter = %.4f ms \n", timer.mvnpdf, max_iter, timer.mvnpdf / max_iter);
		printf("Normalization kernel:             TotTime = %.4f ms; NumIter = %d; TimePerIter = %.4f ms \n", timer.normalization, max_iter, timer.normalization / max_iter);
		printf("Marginal reduction kernel:        TotTime = %.4f ms; NumIter = %d; TimePerIter = %.4f ms \n", timer.marginal_red, max_iter, timer.marginal_red / max_iter);
		printf("Marginal single reduction kernel: TotTime = %.4f ms; NumIter = %d; TimePerIter = %.4f ms \n", timer.marginal_red_sig, max_iter, timer.marginal_red_sig / max_iter);
		printf("Weight update kernel:             TotTime = %.4f ms; NumIter = %d; TimePerIter = %.4f ms \n", timer.weight_update, max_iter, timer.weight_update/ max_iter);
		printf("Mu reduction kernel:              TotTime = %.4f ms; NumIter = %d; TimePerIter = %.4f ms \n", timer.mu_red, max_iter, timer.mu_red_sig/ max_iter);
		printf("Mu single reduction kernel:       TotTime = %.4f ms; NumIter = %d; TimePerIter = %.4f ms \n", timer.mu_red_sig, max_iter, timer.mu_red_sig/ max_iter);
		printf("Sigma reduction kernel:           TotTime = %.4f ms; NumIter = %d; TimePerIter = %.4f ms \n", timer.sigma_red, max_iter, timer.sigma_red / max_iter);
		printf("Sigma single reduction kernel:    TotTime = %.4f ms; NumIter = %d; TimePerIter = %.4f ms \n", timer.sigma_red_sig, max_iter, timer.sigma_red_sig / max_iter);

	}


	/* Copy data from device to host & free device memory */
	// samples
	CUDA_CHECK_RETURN(hipFree(d_samples));

	// likelihood
	CUDA_CHECK_RETURN(hipMemcpy(likelihood, d_likelihood, size_likelihood, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipFree(d_likelihood));
	std::cout << "Likelihood:" << std::endl;
	print_mat(likelihood, num_gaus, s_size);

	// weights
	CUDA_CHECK_RETURN(hipMemcpy(weights, d_weights, size_n_gaus, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipFree(d_weights));
	std::cout << "Weights:" << std::endl;
	print_mat(weights, num_gaus, 1);

	// mu matrix
	CUDA_CHECK_RETURN(hipMemcpy(mu_mat, d_mu_mat, size_mu_mat, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipFree(d_mu_mat));
	std::cout << "Mu mat:" << std::endl;
	print_mat(mu_mat, num_gaus, s_dim);

	// sigma matrix
	CUDA_CHECK_RETURN(hipMemcpy(sig_mat, d_sig_mat, size_sig_mat, hipMemcpyDeviceToHost));
	CUDA_CHECK_RETURN(hipFree(d_sig_mat));
	std::cout << "Sig mat:" << std::endl;
	print_mat(sig_mat, num_gaus, dim_squared);

	// marginals
	CUDA_CHECK_RETURN(hipFree(d_marginals));

	output->mu = mu_mat;
	output->sigma = sig_mat;

	return output[0];
}

int main_no()
{
	int size = sizeof(double) * 3;
	double h_x[3] = {0.3188, -1.3077, -0.4336};
	double h_mu[3] = {0, 0, 0};
	double h_sig[9] = {1, 0, 0, 0, 1, 0, 0, 0, 1};
	double pdf;
	double wpdf;
	double weight = 1 / (double) 3;

	double *d_x, *d_mu, *d_sig, *d_pdf, *d_wpdf, *d_weight;
	hipMalloc((void **)&d_x, size);
	hipMalloc((void **)&d_mu, size);
	hipMalloc((void **)&d_sig, size * 3);
	hipMalloc((void **)&d_pdf, sizeof(double));
	hipMalloc((void **)&d_wpdf, sizeof(double));
	hipMalloc((void **)&d_weight, sizeof(double));

	hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
	hipMemcpy(d_mu, h_mu, size, hipMemcpyHostToDevice);
	hipMemcpy(d_sig, h_sig, size * 3, hipMemcpyHostToDevice);
	hipMemcpy(d_weight, &weight, sizeof(double), hipMemcpyHostToDevice);

	mvnpdf_dim3<<<1, 1>>>(d_pdf, d_x, d_mu, d_sig, 1, 1);

	int shared = sizeof(double) * (3 * 3 + 9 *2);
	w_mvnpdf_dim3<<<1, 1, shared>>>(d_wpdf, d_x, d_mu, d_sig, d_weight, 1, 1);

	hipMemcpy(&pdf, d_pdf, sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(&wpdf, d_wpdf, sizeof(double), hipMemcpyDeviceToHost);

	std::cout << "pdf = " << pdf << std::endl;
	std::cout << "wpdf = " << wpdf << std::endl;

	return 0;
}

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux (const char *file, unsigned line, const char *statement, hipError_t err)
{
	if (err == hipSuccess)
		return;
	std::cerr << statement<<" returned :" << hipGetErrorString(err) << "("<<err<< ") at "<<file<<":"<<line << std::endl;
	exit (1);
}

